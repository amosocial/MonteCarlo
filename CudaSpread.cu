#include "hip/hip_runtime.h"
#include "cudaSpread.cuh"

#include "Header.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

using namespace std;


inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

GpuVs::GpuVs(int b, int t, Network network, vector<int> newActive, bool state[]) : blocks(b), threads(t)
{

	times = b*t;
	d_result = new int[times];


	d_neighbor_size = new int[network.vNum];
	d_neighbor_index = new int[network.vNum];
	d_neighbor = new int[network.eNum];
	d_infos = new int[3];
	d_newActive = new int[newActive.size()];
	d_state = new bool[network.vNum];

	dev_states = new bool[times*network.vNum];
	dev_newActives = new int[times*network.vNum];
	dev_tnewActives = new int[times*network.vNum];

	d_infos[0] = network.vNum; //info_0: number of nodes
	d_infos[1] = newActive.size(); //info_1: num of seed nodes
	d_infos[2] = times; //info_2: times of spread

	//copy current active nodes
	for (size_t i = 0; i < newActive.size(); i++)
	{
		d_newActive[i] = newActive[i];
		//cout << "d_seedSet[i] " << d_seedSet[i] << endl;
	}
	//copy state
	for (size_t i = 0; i < network.vNum; i++)
	{
		d_state[i] = state[i];
	}
	//copy relations
	int count = 0;
	for (size_t i = 0; i < network.vNum; i++)
	{
		//d_neighbor[i] = new int[network.neighbor[i].size()];

		d_neighbor_size[i] = network.neighbor[i].size();
		//cout << d_neighbor_size[i] << endl;
		d_neighbor_index[i] = count;
		for (size_t j = 0; j < d_neighbor_size[i]; j++)
		{
			if (count > network.eNum)
			{
				cout << "count > network.eNum" << endl;
			}
			d_neighbor[count] = network.neighbor[i][j];
			count++;
		}
	}
	//initiate shared records: d_states d_newActives d_tnewActives
	
	hipError_t cudaStatus;

	//----------------------------
	//hipMallocManaged(&d_seedSet, seedSet.size() * sizeof(int));
	//---------------------------------
	// Allocate GPU buffers for four variables  .
	
	
	//-----------------------------------------------
	cudaStatus = hipMalloc((void**)&dev_neighbor, network.eNum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMemcpy(dev_neighbor, d_neighbor, network.eNum * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_neighbor hipMemcpy failed!");
	}

	//-----------------------------------------------
	cudaStatus = hipMalloc((void**)&dev_neighbor_size, network.vNum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMemcpy(dev_neighbor_size, d_neighbor_size, network.vNum * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_neighbor_size hipMemcpy failed!");
	}

	//-----------------------------------------------
	cudaStatus = hipMalloc((void**)&dev_neighbor_index, network.vNum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	cudaStatus = hipMemcpy(dev_neighbor_index, d_neighbor_index, network.vNum * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_neighbor_index hipMemcpy failed!");
	}

	//-----------------------------------------------
	cudaStatus = hipMalloc((void**)&dev_infos, 3 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMemcpy(dev_infos, d_infos, 3 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_infos hipMemcpy failed!");
	}

	//-----------------------------------------------
	cudaStatus = hipMalloc((void**)&dev_state, 3 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMemcpy(dev_state, d_state, 3 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_infos hipMemcpy failed!");
	}
	//-----------------------------------------------
	cudaStatus = hipMalloc((void**)&dev_newActive, 3 * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMemcpy(dev_newActive, d_newActive, 3 * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_infos hipMemcpy failed!");
	}



	//-----------------------------------------------
	cudaStatus = hipMalloc((void**)&dev_result, times * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_states, network.vNum*times * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_newActives, network.vNum*times * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}

	cudaStatus = hipMalloc((void**)&dev_tnewActives, network.vNum*times * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}
	
}






__global__ void initCurand(hiprandState *state, unsigned long seed) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(seed+ idx, 0, 0, &state[idx]);
}

__global__ void marginalKernel(int cseed, int* result, int* neighbor, int* neighbor_size, int* neighbor_index, int* infos, int* newActive, bool* state, bool* states, int* newActives, int* tnewActives, hiprandState* rstates)
{
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	int sindex = id*infos[0];
	int newActive_size = 0;
	int tnewActive_size = 0;

	int aNum = 0;
	for (size_t i = 0; i < infos[0]; i++)
	{
		states[id+i] = false;
		newActives[id+i] = 0;
		tnewActives[id+i] = 0;
	}

	for (size_t i = 0; i <infos[1]; i++)
	{
		states[sindex + newActive[i]] = true;
		newActives[sindex + i] = newActive[i];
		newActive_size++;
		//aNum++;
	}


	while (newActive_size>0)
	{
		for (size_t i = 0; i < newActive_size; i++)
		{
			int seed = newActives[sindex + i];
			//printf("seed %d and size %d \n", seed, d_neighbor_size[seed]);
			for (size_t j = 0; j < neighbor_size[seed]; j++)
			{
				int j_index = neighbor_index[seed] + j;
				int seede = neighbor[j_index];
				//printf("activate %d and %d \n", seede, seede);
				//cout << seed << ' ' << neighbor[seed][j] << endl;


				//hiprand_init((unsigned long long)clock() , id, 0, &s);
				float rand = hiprand_uniform(&rstates[id]);
				//float rand = 0.01;
				float prob = 0.1;//???
								 //printf("rand %f \n", rand);
				if (rand < prob && !states[sindex + seede])
				{
					states[sindex + seede] = true;
					tnewActives[sindex + tnewActive_size] = seede;
					tnewActive_size++;
					//aNum++;
				}
			}
			//cout << tActive.size() << endl;
		}
		//copy new active nodes
		for (size_t i = 0; i < tnewActive_size; i++)
		{
			newActives[sindex + i] = tnewActives[sindex + i];
		}
		newActive_size = tnewActive_size;
		tnewActive_size = 0;
	}

	if (!states[sindex + cseed])
	{
		newActives[sindex + 0] = cseed;
		states[sindex + cseed] = true;
		newActive_size++;
		aNum++;

		while (newActive_size>0)
		{
			for (size_t i = 0; i < newActive_size; i++)
			{
				int seed = newActives[sindex + i];
				for (size_t j = 0; j < neighbor_size[seed]; j++)
				{
					int j_index = neighbor_index[seed] + j;
					int seede = neighbor[j_index];
					float rand = hiprand_uniform(&rstates[id]);
					//float rand = 0.01;
					float prob = 0.1;//???
									 //printf("rand %f \n", rand);
					if (rand < prob && !states[sindex + seede])
					{
						states[sindex + seede] = true;
						tnewActives[sindex + tnewActive_size] = seede;
						tnewActive_size++;
						//aNum++;
					}
				}
				//cout << tActive.size() << endl;
			}
			//copy new active nodes
			for (size_t i = 0; i < tnewActive_size; i++)
			{
				newActives[sindex + i] = tnewActives[sindex + i];
			}
			newActive_size = tnewActive_size;
			tnewActive_size = 0;
		}
	}
	//printf("Hello from thread %d %d----------------------- %d \n", blockIdx.x, threadIdx.x, aNum);

	result[id] = aNum;
}


__global__ void addKernel(int* d_result, int* d_neighbor, int* d_neighbor_size, int* d_neighbor_index, int* d_seedSet, int* infos, bool* states, int* newActives, int* tnewActives, hiprandState* rstates)
{
	
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	int sindex = id*infos[0];
	//printf("Hello from thread %d and %d \n", blockIdx.x, threadIdx.x);
	//bool* state = new bool[infos[0]];
	
	//int* newActive= new int[infos[0]];
	int newActive_size = 0;
	
	//int* tnewActive = new int[infos[0]];
	int tnewActive_size = 0;

	int aNum = 0;
	/*
	for (size_t i = 0; i <infos[0]; i++)
	{
		state[i] = false;
		newActive[i] = 0;
		tnewActive[i] = 0;
	}*/
	
	for (size_t i = 0; i <infos[1]; i++)
	{
		states[sindex+d_seedSet[i]] = true;
		newActives[sindex+i]= d_seedSet[i];
		newActive_size++;
		aNum++;
	}
	
	//hiprandState s;
	//hiprand_init((unsigned long long)clock() +id, 0, 0, &s);
	//hiprandState localState = rstates[id];
	//hiprand_init(id, 0, 0, &localState);
	//for (size_t i = 0; i < 10; i++)
	//{
	//	float rand = hiprand_uniform(&rstates[id]);
	//	printf("rand %f \n", rand);
	//}
	
	while (newActive_size>0)
	{
		for (size_t i = 0; i < newActive_size; i++)
		{
			int seed = newActives[sindex+i];
			//printf("seed %d and size %d \n", seed, d_neighbor_size[seed]);
			for (size_t j = 0; j < d_neighbor_size[seed]; j++)
			{
				int j_index = d_neighbor_index[seed] + j;
				int seede = d_neighbor[j_index];
				//printf("activate %d and %d \n", seede, seede);
				//cout << seed << ' ' << neighbor[seed][j] << endl;

				
				//hiprand_init((unsigned long long)clock() , id, 0, &s);
				float rand = hiprand_uniform(&rstates[id]);
				//float rand = 0.01;
				float prob = 0.1;//???
				//printf("rand %f \n", rand);
				if (rand < prob && !states[sindex+seede])
				{
					states[sindex+seede] = true;
					tnewActives[sindex+tnewActive_size]= seede;
					tnewActive_size++;
					aNum++;
				}
			}
			//cout << tActive.size() << endl;
		}
		//copy new active nodes
		for (size_t i = 0; i < tnewActive_size; i++)
		{
			newActives[sindex+i] = tnewActives[sindex+i];
		}
		newActive_size= tnewActive_size;
		tnewActive_size=0;
	}
	//printf("Hello from thread %d %d----------------------- %d \n", blockIdx.x, threadIdx.x, aNum);
	
	d_result[id] = aNum;

}


hipError_t g_marginal(float &result, Network &network, int cseed, GpuVs gpuvs)
{
	cout << "g_marginal is running " << endl;
	hipError_t cudaStatus;

	hiprandState* rstates;	hipMalloc((void**)&rstates, gpuvs.times * sizeof(hiprandState));
	initCurand << <gpuvs.blocks, gpuvs.threads >> >(rstates, 1);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	marginalKernel << < gpuvs.blocks, gpuvs.threads >> > (cseed,
		gpuvs.dev_result,
		gpuvs.dev_neighbor, 
		gpuvs.dev_neighbor_size,
		gpuvs.dev_neighbor_index,
		gpuvs.dev_infos,
		gpuvs.dev_newActive,
		gpuvs.dev_state,
		gpuvs.dev_states,
		gpuvs.dev_newActives,
		gpuvs.dev_tnewActives,
		rstates);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "marginalKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		getchar();
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		getchar();

	}

	// Copy output vector from GPU buffer to host memory.

	cudaStatus = hipMemcpy(gpuvs.d_result, gpuvs.dev_result, gpuvs.times * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
	}
	//cout << times << endl;
	for (size_t i = 0; i < gpuvs.times; i++)
	{
		//cout << i << " " << gpuvs.d_result[i] << endl;
		result = result + gpuvs.d_result[i];
	}
	// process required information
	
	result = result / gpuvs.times;
	return cudaStatus;

}

// Helper function for using CUDA to add vectors in parallel.

hipError_t spreadWithCuda(vector<int> &result, Network &network, vector<int> &seedSet, int blocks, int threads)
{
	
	

	//blocks and threads
	int times = blocks*threads;

	//setup crandstates
	hiprandState* rstates;	gpuErrchk(hipMalloc((void**)&rstates, times * sizeof(hiprandState)));

	float *d_a;             gpuErrchk(hipMalloc((void**)&d_a, times * sizeof(float)));

	initCurand << <blocks , threads >> >(rstates, 1);
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	//testrand1 << <blocks, threads >> >(rstates, d_a);
	//gpuErrchk(hipPeekAtLastError());
	//gpuErrchk(hipDeviceSynchronize());

	//getchar();
	//network.ShowRelation();
	int* dev_result = 0;
	int* dev_seedSet = 0;

	int* dev_neighbor = 0;
	int* dev_neighbor_size = 0;
	int* dev_neighbor_index = 0;
	int* dev_infos= 0;

	bool* dev_states = 0;
	int* dev_newActives = 0;
	int* dev_tnewActives = 0;

	hipError_t cudaStatus;

	
	// process required information
	int* d_result= new int[times];
	int* d_seedSet = new int[seedSet.size()];

	int* d_neighbor_size = new int[network.vNum];
	int* d_neighbor_index = new int[network.vNum];
	int* d_neighbor = new int[network.eNum];
	int* d_infos = new int[3];

	bool* d_states = new bool[times*network.vNum];
	int* d_newActives = new int[times*network.vNum];
	int* d_tnewActives = new int[times*network.vNum];

	d_infos[0] = network.vNum; //info_0: number of nodes
	d_infos[1] = seedSet.size(); //info_1: num of seed nodes
	d_infos[2] = times; //info_2: times of spread


	for (size_t i = 0; i < seedSet.size(); i++)
	{
		d_seedSet[i] = seedSet[i];
		cout << "d_seedSet[i] " << d_seedSet[i] << endl;
	}
	 
	int count = 0;
	for (size_t i = 0; i < network.vNum; i++)
	{
		//d_neighbor[i] = new int[network.neighbor[i].size()];
		
		d_neighbor_size[i] =  network.neighbor[i].size();
		//cout << d_neighbor_size[i] << endl;
		d_neighbor_index[i] = count;
		for (size_t j = 0; j < d_neighbor_size[i]; j++)
		{
			if (count > network.eNum)
			{
				cout << "count > network.eNum" << endl;
			}
			d_neighbor[count] = network.neighbor[i][j];
			count++;
		}
	}

	for (size_t i = 0; i < times*network.vNum; i++)
	{
		d_states[i] = false;
		d_newActives[i] = 0;
		d_tnewActives[i] = 0;
	}

	

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	//----------------------------
	//hipMallocManaged(&d_seedSet, seedSet.size() * sizeof(int));
	//---------------------------------
	// Allocate GPU buffers for four variables  .
	cudaStatus = hipMalloc((void**)&dev_result, times*sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_seedSet, seedSet.size() * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_neighbor, network.eNum*sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_neighbor_size, network.vNum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_neighbor_index, network.vNum * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_infos, 3*sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_states, network.vNum*times * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_newActives, network.vNum*times * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_tnewActives, network.vNum*times * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	
	cudaStatus = hipMemcpy(dev_seedSet, d_seedSet, seedSet.size() * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_seedSet hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_neighbor, d_neighbor, network.eNum * sizeof(int) , hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_neighbor hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_neighbor_size, d_neighbor_size, network.vNum * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_neighbor_size hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_neighbor_index, d_neighbor_index, network.vNum * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_neighbor_index hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_infos, d_infos, 3*sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_infos hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_states, d_states, network.vNum* times * sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_infos hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_newActives, d_newActives, network.vNum* times * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_infos hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_tnewActives, d_tnewActives, network.vNum* times * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "dev_infos hipMemcpy failed!");
		goto Error;
	}


	//allocate more memory
	/*
	int states_size = network.vNum * sizeof(bool);
	int active_size = network.vNum * sizeof(int);
	int tactive_size = network.vNum * sizeof(int);
	int extra_size = network.vNum * sizeof(int);
	cudaStatus = hipDeviceSetLimit(hipLimitMallocHeapSize, states_size+ active_size+ tactive_size+extra_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSetLimit failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}*/
	//hipMalloc((void**)&rstates, blocks * threads * sizeof(hiprandState));
	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<< blocks, threads >>> (dev_result, dev_neighbor, dev_neighbor_size, dev_neighbor_index, dev_seedSet, dev_infos, dev_states, dev_newActives, dev_tnewActives, rstates);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	
	cudaStatus = hipMemcpy(d_result, dev_result, times*sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//cout << times << endl;
	for (size_t i = 0; i < times; i++)
	{
		cout << i<<" "<<d_result[i] << endl;
	}

Error:
	hipFree(dev_result);
	hipFree(dev_neighbor);
	hipFree(dev_seedSet);
	hipFree(dev_neighbor_size);
	hipFree(dev_neighbor_index);
	hipFree(dev_infos);
	hipFree(dev_states);
	hipFree(dev_newActives);
	hipFree(dev_tnewActives);

	return cudaStatus;
}

